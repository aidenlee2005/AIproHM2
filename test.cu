#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    int a[5] = {1, 2, 3, 4, 5};
    int b[5] = {10, 20, 30, 40, 50};
    int c[5];

    int *d_a, *d_b, *d_c;
    int size = 5 * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<1, 5>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Result: ");
    for(int i = 0; i < 5; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}